#include "hip/hip_runtime.h"
// Conversion from fortran to C++/Cuda still in progressC

unsigned int kxmax,kymax,kzmax,ksmax;
// kxmax must be the largest
unsigned int kdfull;
unsigned int kdred;
unsigned int kfft2;
//INTEGER,PARAMETER,PRIVATE :: kddoub=kdfull
unsigned int kfft,kfftx,kffty,kfftz;
//INTEGER,PARAMETER,PRIVATE :: coulsolv_mp_kdcorf_=(kxmax/2+1)*(kymax/2+1)*(kzmax/2+1);
// include block: xkgrid
//extern double coulsolv_mp_xval_[],coulsolv_mp_yval_[],coulsolv_mp_zval_[];
//extern double coulsolv_mp_xt2_[],coulsolv_mp_yt2_[],coulsolv_mp_zt2_[];
double dx,dy,dz,dxsp,grnorm,fnorm,tnorm;
unsigned int nxc,nyc,nzc,nx1,ny1,nz1;
//unsigned int nxr,nxi,nyr,nyi,nzr,nzi;
unsigned int nxi,nyi,nzi;
unsigned int nxy1,nxyz,nxhigh;
unsigned int nxlow,nyhigh,nylow,nzhigh,nzlow;

//extern int coulsolv_mp_ikm_[][];
double dkx,dky,dkz,akmax,dksp,ecut;
unsigned int nxk,nxklo,nxkhi,nksp,nkxyz;

double zero=0.0;
double pi=3.141592653589793;


//double *akv2r,*akv2i,*pakv2r,*pakv2i;
hipfftHandle pfft;
int batch=1;
hipfftDoubleComplex *fftac,*akvc,*pfftac,*pakvc;
hipfftDoubleComplex *gpu_fftac,*gpu_akvc;
int res;

//-----fourfakv---------------------------------------------------------

void fourfakv(){

//     fourier forward transformation
//     I/O: pskr   real part of the wave-function
//          pski   imaginary part of the wave-function

//DATA  mini/0/              // flag for initialization
//----------------------------------------------------------------------

int blocksize=192;
int gridx=(int)ceil(nxyz/(float)blocksize);
dim3 dimgrid(gridx,1,1);
dim3 dimblock(blocksize,1,1);

//test      sqh=sqrt(0.5)

tnorm=grnorm*fnorm;

hipMemcpyAsync(gpu_akvc,pakvc,kdred*sizeof(hipfftDoubleComplex),hipMemcpyHostToDevice,stream1);
Check_CUDA_Error(error);

if(hipfftExecZ2Z(pfft,gpu_akvc,gpu_akvc, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
{
  cout<<"CUFFT error : Exec Z2Z forward failed in coulex (akv)"<<endl;
  exit(-1);
}

multiply_device<<<dimgrid,dimblock,0,stream2>>>(gpu_akvc,kdred,tnorm);
Check_CUDA_Error(error);

}

//-----fftinp------------------------------------------------------------

void fftinp() {

//     initializes work tables for FFT

//     grid parameters nxc,nyc,nzc,dx,dy,dz,ecut must have been read or
//     initialized before !

//-----------------------------------------------------------------------

int ikzero,ii;
double xz1,xz2,xy1,xy2,xx1,xx2,ak2;
int nxyfn,ind;

//test      sqh=sqrt(0.5)

//     initialize grid in coordinate space

nx1=nxc+1;
ny1=nyc+1;
nz1=nzc+1;
nxi=nxc+nxc;
nyi=nyc+nyc;
nzi=nzc+nzc;
nxy1=nxi*nyi;
nxyz=nxi*nyi*nzi;
nkxyz=nxi*nyi*nzi;

nxyfn = kfftx*kfftz;

//     grid lengths must match with parameters in incs

if(kxmax < nxi) {
  cout<< "ERROR: parameter   kxmax   too small"<<endl;
  cout<< " error in parameter: KXMAX in COULEX too small"<<endl;
  exit(-1);
}
else if(kymax < nyi) {
  cout<< " ERROR: parameter   kymax   too small"<<endl;
  cout<< " error in parameter: KYMAX in COULEX too small"<<endl;
  exit(-1);
}
else if(kzmax < nzi) {
  cout<< " ERROR: parameter   kzmax   too small"<<endl;
  cout<< " error in parameter: KZMAX in COULEX too small"<<endl;
  exit(-1);
}

//     initialize grid in fourier space

dkx=pi/(dx*double(nxc));
dky=pi/(dy*double(nyc));
dkz=pi/(dz*double(nzc));

dxsp=dx*dy*dz;
dksp=dkx*dky*dkz;
cout<< " dkx,dky,dkz,dksp= "<<dkx<<" "<<dky<<" "<<dkz<<" "<<dksp<<endl;

grnorm=sqrt(dxsp/dksp);
fnorm=1.0/sqrt(double(nxc*nyc*nzc));
nxk=nx1;

//     built Greens function in Fourier space
//     by Fourier transformation from real space

ikzero = nxy1*(nzc-1)+nxi*(nyc-1)+nxc;
cout<< " nzi,nyi,nxi,nx,ny,nz,ikzero= "<<nzi<<" "<<nyi<<" "<<nxi<<" "<<nxc<<" "<<nyc<<" "<<nzc<<" "<<ikzero<<endl;
ii=0;
xz1=-(1.0)*nzc*dz;
for (int i3=1;i3<=nzi;i3++){
  xz1=xz1+dz;
  xz2=xz1*xz1;
  xy1=-(1.0)*nyc*dy;
  for (int i2=1;i2<=nyi;i2++){
    xy1=xy1+dy;
    xy2=xy1*xy1;
    xx1=-(1.0)*nxc*dx;
    for (int i1=1;i1<=nxi;i1++){
      xx1=xx1+dx;
      xx2=xx1*xx1;
      ak2=xx2+xy2+xz2;
      ii=ii+1;
//        cout<< " i1,i2,i3,ii= "<<i1<<" "<<i2<<" "<<i3<<" "ii;
      ind=((i3+nxc)%nxi)*nxyfn+((i2+nyc)%nyi)*kfftx+(i1+nyc)%nzi+1; //storage in a flatten 3D complex array for FFT on GPU
      if(ii != ikzero) {
        akvc[ind].x =  1.0/sqrt(ak2);
      }
      else {
//              akv2r(ii) = (6D0*pi/(dx*dy*dz))**(1D0/3D0)  // spherical approx
//              akv2r(ii) = 1.19003868*(dx*dy*dz)**(-1D0/3D0)
        akvc[ind].x = 2.34*1.19003868*pow((dx*dy*dz),(-1.0/3.0));  // empirical
      }
      akvc[ind].y = 0.0;
    }
  }
}
nksp=ii;

fourfakv();

hipHostFree(pakvc);

}

//-------------------------------------------------------------------

extern "C" void init_coul_(double *dx0,double *dy0,double *dz0,unsigned int *nx0,unsigned int *ny0,unsigned int *nz0) {

//-----------------------------------------------------------------------


//     read grid parameters from file or simply initialize them
//     note that the Coulomb solver doubles the grid internally
nxc=*nx0;  ///2;
nyc=*ny0;  ///2;
nzc=*nz0;  ///2;
dx=*dx0;
dy=*dy0;
dz=*dz0;

kxmax=2*nxc;kymax=2*nyc;kzmax=2*nzc;ksmax=kxmax;
kdfull=nxc*nyc*nzc;
kdred=kxmax*kymax*kzmax;
kfft=ksmax;kfftx=kxmax;kffty=kymax;kfftz=kzmax;
kfft2=kfft*2;

//     check initialization

if(hipfftPlan3d(&pfft,kxmax,kymax,kzmax,HIPFFT_Z2Z) != HIPFFT_SUCCESS)
{
  cout<<"CUFFT error : Plan Creation failed"<<endl;
  exit(-1);
}
if(hipfftSetStream(pfft,stream2) != HIPFFT_SUCCESS)
{
  cout<<"CUFFT error : Streamed FFT Creation failed"<<endl;
  exit(-1);
}

// Pinned memory allocation on the CPU to make CPU>GPU and GPU>CPU transfers faster

hipHostMalloc (&pfftac,kdred*sizeof(hipfftDoubleComplex));
hipHostMalloc (&pakvc,kdred*sizeof(hipfftDoubleComplex));

fftac=pfftac-1; //fftac points one location before pfftac, so fftac[1]...fftac[kdred] all exist (just sticks with the fortran convention)
akvc=pakvc-1;   //same trick as above

// Memory allocation on the GPU
hipMalloc((void**)&gpu_fftac,kdred*sizeof(hipfftDoubleComplex));
hipMalloc((void**)&gpu_akvc,kdred*sizeof(hipfftDoubleComplex));

//     call input routine fftinp, which initializes the grid and fft tabl

fftinp();
}

//-----fft--------------------------------------------------------------

void fft() {

int blocksize=192;
int gridx=(int)ceil(nxyz/(float)blocksize);
dim3 dimgrid(gridx,1,1);
dim3 dimblock(blocksize,1,1);

tnorm=grnorm*fnorm;

hipMemcpyAsync(gpu_fftac,pfftac,kdred*sizeof(hipfftDoubleComplex),hipMemcpyHostToDevice,stream1);
Check_CUDA_Error(error);

if(hipfftExecZ2Z(pfft,gpu_fftac,gpu_fftac, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
{
  cout<<"CUFFT error : Exec Z2Z forward failed in coulex"<<endl;
  exit(-1);
}

multiply_device<<<dimgrid,dimblock,0,stream2>>>(gpu_fftac,nxyz,tnorm);
Check_CUDA_Error(error);

}

//-----ffb--------------------------------------------------------------

void ffb() {

//----------------------------------------------------------------------

int blocksize=192;
int gridx=(int)ceil(nxyz/(float)blocksize);
dim3 dimgrid(gridx,1,1);
dim3 dimblock(blocksize,1,1);

tnorm=fnorm/(8.0*grnorm)*pow(pi,1.5);

if(hipfftExecZ2Z(pfft,gpu_fftac,gpu_fftac, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
{
	  cout<<"CUFFT error : Exec Z2Z backward failed in coulex"<<endl;
	  exit(-1);
}

multiply_device<<<dimgrid,dimblock,0,stream2>>>(gpu_fftac,kdred,tnorm);
Check_CUDA_Error(error);

hipMemcpy(pfftac,gpu_fftac,kdred*sizeof(hipfftDoubleComplex),hipMemcpyDeviceToHost);
Check_CUDA_Error(error);

}

//-----cofows------------------------------------------------------------

void coufou2(){

int blocksize=192;
int gridx=(int)ceil(nxyz/(float)blocksize);
dim3 dimgrid(gridx,1,1);
dim3 dimblock(blocksize,1,1);

//------------------------------------------------------------------------------

//     fourier transformation of the density

fft();

//     calculation of the coulomb field (writing on the density field)

multiply_ak_gpu<<<dimgrid,dimblock,0,stream2>>>(gpu_fftac,gpu_akvc,kdred);
Check_CUDA_Error(error);

//     fourier back transformation

ffb();

}

//-----rhofld------------------------------------------------------------

void rhofld(double *rhoinp){

//     copy density on complex array of double extnesion in x,y,z

int nxyfn,nyfn,nnx2,nny2,nnz2,i0,ii;

nxyfn = kfftx*kfftz;
nyfn  = kfftx;
nnx2=nxc+nxc;
nny2=nyc+nyc;
nnz2=nzc+nzc;

i0=0;
for (int i3=1;i3<=kfftz;i3++){
  for (int i2=1;i2<=kffty;i2++){
    for (int i1=1;i1<=kfftx;i1++){
      ii=((i3+nxc)%nnx2)*nxyfn+((i2+nyc)%nny2)*nyfn+(i1+nzc)%nnz2+1;
      if(i3 <= nzc && i2 <= nyc && i1 <= nxc) {
        i0 = i0+1;
        fftac[ii].x=rhoinp[i0];
      }
      else fftac[ii].x=0.0;
      fftac[ii].y=0.0;
    }
  }
}

}


//-----result------------------------------------------------------------

void result(double *chpfalr){

//     copy Coulomb field back to standard grid
int nxyfn,nyfn,nnx2,nny2,nnz2,ii;

int i0=0;

nxyfn = kfftx*kfftz;
nyfn  = kfftx;
nnx2=nxc+nxc;
nny2=nyc+nyc;
nnz2=nzc+nzc;

for (int i3=1;i3<=nzc;i3++){
  for (int i2=1;i2<=nyc;i2++){
    for (int i1=1;i1<=nxc;i1++){
        i0++;
        ii=((i3+nxc)%nnx2)*nxyfn+((i2+nyc)%nny2)*nyfn+(i1+nzc)%nnz2+1;
        chpfalr[i0] = 2.0*fftac[ii].x;
    }
  }
}

}

//-------------------------------------------------------------------

extern "C" void falr_(double *prhoinp,double *pchpfalr,int nxdum,int nydum,int nzdum,int kdum) {

double *rhoinp,*chpfalr;

rhoinp=prhoinp-1; //rhoinp points one location before pfftac, so rhoinp[1]...rhoinp[kdred] all exist (just sticks with the fortran convention)
chpfalr=pchpfalr-1; //same trick as above

//     call a routine written by you which writes your density field
//     on the array rho.
//     remember not to send your original density array to the fcs.
//     in this case we have a homogeneously charged sphere .

rhofld(rhoinp);

//     call coufou, which contains the fcs procedure.
coufou2();

//     call a routine written by you which outputs the results of the fcs
//     and maybe some other things to an output file or the screen.

result(chpfalr);

}

extern "C" void coulsolv_end_() {

hipHostFree(pfftac);
hipFree(gpu_fftac);
hipFree(gpu_akvc);
hipfftDestroy(pfft);

}
