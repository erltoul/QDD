#include "hip/hip_runtime.h"
// Conversion from fortran to C++/Cuda still in progressC

unsigned int kxmax,kymax,kzmax,ksmax;
// kxmax must be the largest
unsigned int kdfull;
unsigned int kdred;
unsigned int kfft2;
//INTEGER,PARAMETER,PRIVATE :: kddoub=kdfull
unsigned int kfft,kfftx,kffty,kfftz;
//INTEGER,PARAMETER,PRIVATE :: coulsolv_mp_kdcorf_=(kxmax/2+1)*(kymax/2+1)*(kzmax/2+1);
// include block: xkgrid
//extern double coulsolv_mp_xval_[],coulsolv_mp_yval_[],coulsolv_mp_zval_[];
//extern double coulsolv_mp_xt2_[],coulsolv_mp_yt2_[],coulsolv_mp_zt2_[];
double dx,dy,dz,dxsp,grnorm,fnorm,tnorm;
unsigned int nxc,nyc,nzc,nx1,ny1,nz1;
//unsigned int nxr,nxi,nyr,nyi,nzr,nzi;
unsigned int nxi,nyi,nzi;
unsigned int nxy1,nxyz,nxhigh;
unsigned int nxlow,nyhigh,nylow,nzhigh,nzlow;

//extern int coulsolv_mp_ikm_[][];
double dkx,dky,dkz,akmax,dksp,ecut;
unsigned int nxk,nxklo,nxkhi,nksp,nkxyz;

double zero=0.0;
double pi=3.141592653589793;


double *akv2r,*akv2i,*pakv2r,*pakv2i;
hipfftHandle pfft;
int batch=1;
hipfftDoubleComplex *fftac,*akvc,*pfftac,*pakvc;
hipfftDoubleComplex *gpu_fftac,*gpu_akvc;
int res;

//-----fourfakv---------------------------------------------------------

void fourfakv(double *pskr,double *pski){

//     fourier forward transformation
//     I/O: pskr   real part of the wave-function
//          pski   imaginary part of the wave-function

//DATA  mini/0/              // flag for initialization
//----------------------------------------------------------------------

int blocksize=192;
int gridx=(int)ceil(nxyz/(float)blocksize);
dim3 dimgrid(gridx,1,1);
dim3 dimblock(blocksize,1,1);

//int nxyfn,nyfn,nnx2,nny2,nnz2,ind1,ind2;

//test      sqh=sqrt(0.5)

tnorm=grnorm*fnorm;

/*nxyfn = kfftx*kfftz;
nyfn  = kfftx;
nnx2=nxc+nxc;
nny2=nyc+nyc;
nnz2=nzc+nzc;*/

/*ind1=0;

for (int i3=1;i3<=kfftz;i3++){
  for (int i2=1;i2<=kffty;i2++){
    for (int i1=1;i1<=kfftx;i1++){
      ind1++;
      ind2=((i3+nxc)%nnx2)*nxyfn+((i2+nyc)%nny2)*nyfn+(i1+nyc)%nnz2+1;
      akvc[ind2].x=pskr[ind1];
      akvc[ind2].y=pski[ind1];
    }
  }
}*/

/*FILE * pFile;
pFile = fopen ("Test","w");
for (int ii=0;ii<kdred;ii++){
  fprintf(pFile,"%d\t%1.10e\t%1.10e\n",ii,pakvc[ii].x,pakvc[ii].y);
}
fclose(pFile);
exit(-1);*/

hipMemcpyAsync(gpu_akvc,pakvc,kdred*sizeof(hipfftDoubleComplex),hipMemcpyHostToDevice,stream1);
Check_CUDA_Error(error);

if(hipfftExecZ2Z(pfft,gpu_akvc,gpu_akvc, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
{
  cout<<"CUFFT error : Exec Z2Z forward failed in coulex (akv)"<<endl;
  exit(-1);
}

multiply_device<<<dimgrid,dimblock,0,stream2>>>(gpu_akvc,kdred,tnorm);
Check_CUDA_Error(error);

}

//-----fftinp------------------------------------------------------------

void fftinp() {

//     initializes work tables for FFT

//     grid parameters nxc,nyc,nzc,dx,dy,dz,ecut must have been read or
//     initialized before !

//-----------------------------------------------------------------------

int ikzero,ii;
double xz1,xz2,xy1,xy2,xx1,xx2,ak2;
int nxyfn,ind;//,nyfn,nnx2,nny2,nnz2,ind1,ind2;

//test      sqh=sqrt(0.5)

//     initialize grid in coordinate space

nx1=nxc+1;
ny1=nyc+1;
nz1=nzc+1;
nxi=nxc+nxc;
nyi=nyc+nyc;
nzi=nzc+nzc;
nxy1=nxi*nyi;
nxyz=nxi*nyi*nzi;
nkxyz=nxi*nyi*nzi;

nxyfn = kfftx*kfftz;

//     grid lengths must match with parameters in incs

if(kxmax < nxi) {
  cout<< "ERROR: parameter   kxmax   too small"<<endl;
  cout<< " error in parameter: KXMAX in COULEX too small"<<endl;
  exit(-1);
}
else if(kymax < nyi) {
  cout<< " ERROR: parameter   kymax   too small"<<endl;
  cout<< " error in parameter: KYMAX in COULEX too small"<<endl;
  exit(-1);
}
else if(kzmax < nzi) {
  cout<< " ERROR: parameter   kzmax   too small"<<endl;
  cout<< " error in parameter: KZMAX in COULEX too small"<<endl;
  exit(-1);
}

//     initialize grid in fourier space

dkx=pi/(dx*double(nxc));
dky=pi/(dy*double(nyc));
dkz=pi/(dz*double(nzc));

dxsp=dx*dy*dz;
dksp=dkx*dky*dkz;
cout<< " dkx,dky,dkz,dksp= "<<dkx<<" "<<dky<<" "<<dkz<<" "<<dksp<<endl;

grnorm=sqrt(dxsp/dksp);
fnorm=1.0/sqrt(double(nxc*nyc*nzc));
nxk=nx1;

//     built Greens function in Fourier space
//     by Fourier transformation from real space

ikzero = nxy1*(nzc-1)+nxi*(nyc-1)+nxc;
cout<< " nzi,nyi,nxi,nx,ny,nz,ikzero= "<<nzi<<" "<<nyi<<" "<<nxi<<" "<<nxc<<" "<<nyc<<" "<<nzc<<" "<<ikzero<<endl;
ii=0;
xz1=-(1.0)*nzc*dz;
for (int i3=1;i3<=nzi;i3++){
  xz1=xz1+dz;
  xz2=xz1*xz1;
  xy1=-(1.0)*nyc*dy;
  for (int i2=1;i2<=nyi;i2++){
    xy1=xy1+dy;
    xy2=xy1*xy1;
    xx1=-(1.0)*nxc*dx;
    for (int i1=1;i1<=nxi;i1++){
      xx1=xx1+dx;
      xx2=xx1*xx1;
      ak2=xx2+xy2+xz2;
      ii=ii+1;
//        cout<< " i1,i2,i3,ii= "<<i1<<" "<<i2<<" "<<i3<<" "ii;
      ind=((i3+nxc)%nxi)*nxyfn+((i2+nyc)%nyi)*kfftx+(i1+nyc)%nzi+1; //storage in a flatten 3D complex array for FFT on GPU
      if(ii != ikzero) {
        //akv2r[ii] =  1.0/sqrt(ak2);
        akvc[ind].x =  1.0/sqrt(ak2);
      }
      else {
//              akv2r(ii) = (6D0*pi/(dx*dy*dz))**(1D0/3D0)  // spherical approx
//              akv2r(ii) = 1.19003868*(dx*dy*dz)**(-1D0/3D0)
        //akv2r[ii] = 2.34*1.19003868*pow((dx*dy*dz),(-1.0/3.0));  // empirical
        akvc[ind].x = 2.34*1.19003868*pow((dx*dy*dz),(-1.0/3.0));  // empirical
      }
      //akv2i[ii] = 0.0;
      akvc[ind].y = 0.0;
    }
  }
}
nksp=ii;

/*FILE * pFile;
pFile = fopen ("Test","w");
for (int ii=1;ii<=kdred;ii++){
  fprintf(pFile,"%d\t%1.10e\t%1.10e\n",ii,akv2r[ii],akv2i[ii]);
}
fclose(pFile);
exit(-1);*/

fourfakv(&akv2r[0],&akv2i[0]);

/*hipMemcpy(pakvc,gpu_akvc,kdred*sizeof(hipfftDoubleComplex),hipMemcpyDeviceToHost);
Check_CUDA_Error(error);

FILE * pFile;
pFile = fopen ("Test","w");
for (int ii=1;ii<=kdred;ii++){
  fprintf(pFile,"%d\t%1.10e\t%1.10e\n",ii,akvc[ii].x,akvc[ii].y);
}
fclose(pFile);
exit(-1);*/
hipHostFree(pakvc);
free(pakv2r);
free(pakv2i);       //Only gpu_akvc will be used from now

}

//-------------------------------------------------------------------

extern "C" void init_coul_(double *dx0,double *dy0,double *dz0,unsigned int *nx0,unsigned int *ny0,unsigned int *nz0) {

//-----------------------------------------------------------------------


//     read grid parameters from file or simply initialize them
//     note that the Coulomb solver doubles the grid internally
nxc=*nx0;  ///2;
nyc=*ny0;  ///2;
nzc=*nz0;  ///2;
dx=*dx0;
dy=*dy0;
dz=*dz0;

kxmax=2*nxc;kymax=2*nyc;kzmax=2*nzc;ksmax=kxmax;
kdfull=nxc*nyc*nzc;
kdred=kxmax*kymax*kzmax;
kfft=ksmax;kfftx=kxmax;kffty=kymax;kfftz=kzmax;
kfft2=kfft*2;

//     check initialization

if(hipfftPlan3d(&pfft,kxmax,kymax,kzmax,HIPFFT_Z2Z) != HIPFFT_SUCCESS)
{
  cout<<"CUFFT error : Plan Creation failed"<<endl;
  exit(-1);
}
if(hipfftSetStream(pfft,stream2) != HIPFFT_SUCCESS)
{
  cout<<"CUFFT error : Streamed FFT Creation failed"<<endl;
  exit(-1);
}

//pakv2r = (double *) calloc(kdred,sizeof(double));
//pakv2i = (double *) calloc(kdred,sizeof(double));

//akv2r=pakv2r-1;
//akv2i=pakv2i-1;

// Pinned memory allocation on the CPU to make CPU>GPU and GPU>CPU transfers faster

hipHostMalloc (&pfftac,kdred*sizeof(hipfftDoubleComplex));
hipHostMalloc (&pakvc,kdred*sizeof(hipfftDoubleComplex));

fftac=pfftac-1; //fftac points one location before pfftac, so fftac[1]...fftac[kdred] all exist (just sticks with the fortran convention)
akvc=pakvc-1;   //same trick as above

// Memory allocation on the GPU
hipMalloc((void**)&gpu_fftac,kdred*sizeof(hipfftDoubleComplex));
hipMalloc((void**)&gpu_akvc,kdred*sizeof(hipfftDoubleComplex));

//     call input routine fftinp, which initializes the grid and fft tabl

fftinp();
}

//-----fft--------------------------------------------------------------

void fft(double *psxr,double *psxi) {

int blocksize=192;
int gridx=(int)ceil(nxyz/(float)blocksize);
dim3 dimgrid(gridx,1,1);
dim3 dimblock(blocksize,1,1);

int nxyfn,nyfn,nnx2,nny2,nnz2,ind1,ind2;
//int nxyfn,nyfn,ind1,ind2;

tnorm=grnorm*fnorm;

//copyr1dto3d(&psxr,&psxi,*fftac,kfftx,kffty,kfftz);

nxyfn = kfftx*kfftz;
nyfn  = kfftx;
nnx2=nxc+nxc;
nny2=nyc+nyc;
nnz2=nzc+nzc;

ind1=0;
  for (int i3=1;i3<=kfftz;i3++){
    for (int i2=1;i2<=kffty;i2++){
      for (int i1=1;i1<=kfftx;i1++){
      ind1++;
      ind2=((i3+nxc)%nnx2)*nxyfn+((i2+nyc)%nny2)*nyfn+(i1+nzc)%nnz2+1;
      fftac[ind2].x=psxr[ind1];
      fftac[ind2].y=psxi[ind1];
    }
  }
}

/*FILE * pFile;
pFile = fopen ("Test","w");
for (int ii=0;ii<kdred;ii++){
  fprintf(pFile,"%d\t%1.10e\t%1.10e\n",ii,pfftac[ii].x,pfftac[ii].y);
}
fclose(pFile);
exit(-1);*/

hipMemcpyAsync(gpu_fftac,pfftac,kdred*sizeof(hipfftDoubleComplex),hipMemcpyHostToDevice,stream1);
Check_CUDA_Error(error);

if(hipfftExecZ2Z(pfft,gpu_fftac,gpu_fftac, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
{
  cout<<"CUFFT error : Exec Z2Z forward failed in coulex"<<endl;
  exit(-1);
}
/*hipMemcpy(fftac,gpu_fftac,kdred*sizeof(hipfftDoubleComplex),hipMemcpyDeviceToHost);
Check_CUDA_Error(error);
FILE * pFile;
pFile = fopen ("Test","w");
for (int ii=0;ii<kdred;ii++){
  fprintf(pFile,"%d\t%1.10e\t%1.10e\n",ii,fftac[ii].x,fftac[ii].y);
}
fclose(pFile);
exit(-1);*/

multiply_device<<<dimgrid,dimblock,0,stream2>>>(gpu_fftac,nxyz,tnorm);
Check_CUDA_Error(error);

}

//-----ffb--------------------------------------------------------------

void ffb(double *psxr,double *psxi) {

//----------------------------------------------------------------------

int blocksize=192;
int gridx=(int)ceil(nxyz/(float)blocksize);
dim3 dimgrid(gridx,1,1);
dim3 dimblock(blocksize,1,1);

int nxyfn,nyfn,nnx2,nny2,nnz2,ind1,ind2;

tnorm=fnorm/(8.0*grnorm)*pow(pi,1.5);

if(hipfftExecZ2Z(pfft,gpu_fftac,gpu_fftac, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
{
	  cout<<"CUFFT error : Exec Z2Z backward failed in coulex"<<endl;
	  exit(-1);
}

multiply_device<<<dimgrid,dimblock,0,stream2>>>(gpu_fftac,kdred,tnorm);
Check_CUDA_Error(error);

hipMemcpy(pfftac,gpu_fftac,kdred*sizeof(hipfftDoubleComplex),hipMemcpyDeviceToHost);
Check_CUDA_Error(error);

nxyfn = kfftx*kfftz;
nyfn  = kfftx;
nnx2=nxc+nxc;
nny2=nyc+nyc;
nnz2=nzc+nzc;

ind1=0;
  for (int i3=1;i3<=kfftz;i3++){
    for (int i2=1;i2<=kffty;i2++){
      for (int i1=1;i1<=kfftx;i1++){
      ind1++;
      ind2=((i3+nxc)%nnx2)*nxyfn+((i2+nyc)%nny2)*nyfn+(i1+nzc)%nnz2+1;
//      psxr[ind]=fftac[(i1+nnx2)%kfftx+1][(i2+nny2)%kffty+1][(i3+nnz2)%kfftz+1].x;
//      psxi[ind]=fftac[(i1+nnx2)%kfftx+1][(i2+nny2)%kffty+1][(i3+nnz2)%kfftz+1].y;
      psxr[ind1]=fftac[ind2].x;
      psxi[ind1]=fftac[ind2].y;
    }
  }
}

/*FILE * pFile;
pFile = fopen ("Test","w");
for (int ii=1;ii<=kdred;ii++){
  fprintf(pFile,"%d\t%1.10e\t%1.10e\n",ii,fftac[ii].x,fftac[ii].y);
}
fclose(pFile);
exit(-1);*/

//copyr3dto1d(*fftac,&psxr,&psxi,kfftx,kffty,kfftz);

}

//-----cofows------------------------------------------------------------

void coufou2(double *rhokr,double *rhoki){

int blocksize=192;
int gridx=(int)ceil(nxyz/(float)blocksize);
dim3 dimgrid(gridx,1,1);
dim3 dimblock(blocksize,1,1);

//------------------------------------------------------------------------------

//     fourier transformation of the density

/*FILE * pFile;
pFile = fopen ("Test","w");
for (int ii=0;ii<kdred;ii++){
  fprintf(pFile,"%d\t%1.10e\t%1.10e\n",ii,rhokr[ii],rhoki[ii]);
}
fclose(pFile);
exit(-1);*/

fft(rhokr,rhoki);

//     calculation of the coulomb field (writing on the density field)

multiply_ak_gpu<<<dimgrid,dimblock,0,stream2>>>(gpu_fftac,gpu_akvc,kdred);
Check_CUDA_Error(error);

//     fourier back transformation

ffb(rhokr,rhoki);

}

//-----rhofld------------------------------------------------------------

void rhofld(double *rhoinp,double *rhokr,double *rhoki){

//     copy density on complex array of double extnesion in x,y,z

int i0,ii;

i0=0;
//FILE * pFile;
//pFile = fopen ("Test","w");
for (int i3=1;i3<=nzc;i3++){
  for (int i2=1;i2<=nyc;i2++){
    ii = (i3-1)*nxi*nyi+(i2-1)*nxi;
    for (int i1=1;i1<=nxc;i1++){
      ii=ii+1;
      i0 = i0+1;
      rhokr[ii]=rhoinp[i0];
      rhoki[ii]=0.0;
//      fprintf(pFile,"%d\t%1.10e\t%d\t%1.10e\n",ii,rhokr[ii],i0,rhoinp[i0]);
    }
  }
}
//fclose(pFile);
//exit(-1);
}


//-----result------------------------------------------------------------

void result(double *chpfalr,double *rhokr){

//     copy Coulomb field back to standard grid

int ii=0;
int i0=0;
//FILE * pFile;
//pFile = fopen ("Test","w");
for (int i3=1;i3<=nzi;i3++){
  for (int i2=1;i2<=nyi;i2++){
    for (int i1=1;i1<=nxi;i1++){
      ii++;
      if(i3 <= nzc && i2 <= nyc && i1 <= nxc) {
        i0++;
        chpfalr[i0] = 2.0*rhokr[ii];
	//fprintf(pFile,"%d\t%lf\t%d\t%lf\n",i0,chpfalr[i0],ii,rhokr[ii]);
      }
    }
  }
}
//fclose(pFile);
}

//-------------------------------------------------------------------

extern "C" void falr_(double *prhoinp,double *pchpfalr,int nxdum,int nydum,int nzdum,int kdum) {

double *rhokr,*rhoki,*prhokr,*prhoki,*rhoinp,*chpfalr;

prhokr = (double *) calloc(kdred,sizeof(double));
prhoki = (double *) calloc(kdred,sizeof(double));

rhokr=prhokr-1;
rhoki=prhoki-1;

rhoinp=prhoinp-1;
chpfalr=pchpfalr-1;

//     call a routine written by you which writes your density field
//     on the array rho.
//     remember not to send your original density array to the fcs.
//     in this case we have a homogeneously charged sphere .


rhofld(rhoinp,rhokr,rhoki);


//     call coufou, which contains the fcs procedure.
/*FILE * pFile;
pFile = fopen ("Test","w");
for (int ii=0;ii<kdred;ii++){
  fprintf(pFile,"%d\t%1.10e\t%1.10e\n",ii,rhokr[ii],rhoki[ii]);
}
fclose(pFile);
exit(-1);*/
coufou2(rhokr,rhoki);

/*FILE * pFile;
pFile = fopen ("Test","w");
for (int ii=0;ii<kdred;ii++){
  fprintf(pFile,"%d\t%1.10e\t%1.10e\n",ii,rhokr[ii],rhoki[ii]);
}
fclose(pFile);
exit(-1);*/

//     call a routine written by you which outputs the results of the fcs
//     and maybe some other things to an output file or the screen.

result(chpfalr,rhokr);

free(prhokr);
free(prhoki);

}

extern "C" void coulsolv_end_() {

hipHostFree(pfftac);
hipFree(gpu_fftac);
hipFree(gpu_akvc);
hipfftDestroy(pfft);

}
