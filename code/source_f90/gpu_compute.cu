#include "hip/hip_runtime.h"
//All sum and multiplication routines on GPU

__global__ void multiply_device(hipfftDoubleComplex *d_ffta,int nxyz,double norm)
{
	unsigned int ind = blockIdx.x*blockDim.x+threadIdx.x;

	if (ind<nxyz)
	{
		d_ffta[ind].x=norm*d_ffta[ind].x;
		d_ffta[ind].y=norm*d_ffta[ind].y;
	}
}

__global__ void multiply_device_real(hipfftDoubleReal *d_ffta,int nxyz,double norm)
{
	unsigned int ind = blockIdx.x*blockDim.x+threadIdx.x;

	if (ind<nxyz)
	{
		d_ffta[ind]=norm*d_ffta[ind];
	}
}

extern "C" void multiply_gpu_(hipfftDoubleComplex *d_ffta,int *N,double *tnorm)
{
	int nxyz = *N;
	double norm = *tnorm;
	int blocksize=192;
	int gridx=(int)ceil(nxyz/(float)blocksize);
	dim3 dimgrid(gridx,1,1);
	dim3 dimblock(blocksize,1,1);

	//Multiplication d_ffta*norm on the GPU
        multiply_device<<<dimgrid,dimblock,0,stream2>>>(d_ffta,nxyz,norm);
	Check_CUDA_Error(error);
}

__global__ void multiply_ak_gpu(hipfftDoubleComplex *d_ffta,hipfftDoubleComplex *d_ak,int nxyz, double tnorm)
{
	unsigned int ind = blockIdx.x*blockDim.x+threadIdx.x;
	hipfftDoubleReal SAVE2;

	if (ind<nxyz)
	{
		SAVE2         = d_ak[ind].x*d_ffta[ind].x+d_ak[ind].y*d_ffta[ind].y;
		d_ffta[ind].y = (d_ak[ind].x*d_ffta[ind].y+d_ak[ind].y*d_ffta[ind].x)*tnorm;
		d_ffta[ind].x = SAVE2*tnorm;
	}
}

extern "C" void multiply_ak_(hipfftDoubleComplex *d_ffta,hipfftDoubleComplex *d_ak,int *N)
{
	int nxyz = *N;
	int blocksize=192;
	int gridx=(int)ceil(nxyz/(float)blocksize);
	dim3 dimgrid(gridx,1,1);
	dim3 dimblock(blocksize,1,1);

	//Multiplication d_ffta*d_ak on the GPU
        multiply_ak_gpu<<<dimgrid,dimblock,0,stream2>>>(d_ffta,d_ak,nxyz,1.0);
	Check_CUDA_Error(error);
}

__global__ void multiply_ak_gpu2(hipfftDoubleComplex *d_ffta,hipfftDoubleComplex *d_ak,int nxyz)
{
	unsigned int ind = blockIdx.x*blockDim.x+threadIdx.x;
	hipfftDoubleReal SAVE2;

	if (ind<nxyz)
	{
		SAVE2         = d_ak[ind].x*d_ffta[ind].x-d_ak[ind].y*d_ffta[ind].y;
		d_ffta[ind].y = d_ak[ind].x*d_ffta[ind].y+d_ak[ind].y*d_ffta[ind].x;
		d_ffta[ind].x = SAVE2;
	}
}

extern "C" void multiply_ak2_(hipfftDoubleComplex *d_ffta,hipfftDoubleComplex *d_ak,int *N)
{
	int nxyz = *N;
	int blocksize=192;
	int gridx=(int)ceil(nxyz/(float)blocksize);
	dim3 dimgrid(gridx,1,1);
	dim3 dimblock(blocksize,1,1);

	//Multiplication d_ffta*d_ak on the GPU
        multiply_ak_gpu2<<<dimgrid,dimblock,0,stream2>>>(d_ffta,d_ak,nxyz);
	Check_CUDA_Error(error);
}

__global__ void multiply_ak_real_gpu(hipfftDoubleComplex *d_ffta,hipfftDoubleReal *d_ak,int nxyz)
{
	unsigned int ind = blockIdx.x*blockDim.x+threadIdx.x;

	if (ind<nxyz)
	{
		d_ffta[ind].x = d_ak[ind]*d_ffta[ind].x;
		d_ffta[ind].y = d_ak[ind]*d_ffta[ind].y;
	}
}

extern "C" void multiply_ak_real_(hipfftDoubleComplex *d_ffta,hipfftDoubleReal *d_ak,int *N)
{
	int nxyz = *N;
	int blocksize=192;
	int gridx=(int)ceil(nxyz/(float)blocksize);
	dim3 dimgrid(gridx,1,1);
	dim3 dimblock(blocksize,1,1);

	//Multiplication d_ffta*d_ak on the GPU
        multiply_ak_real_gpu<<<dimgrid,dimblock,0,stream2>>>(d_ffta,d_ak,nxyz);
	Check_CUDA_Error(error);
}

__global__ void multiply_shift_gpu(hipfftDoubleComplex *d_ffta,hipfftDoubleComplex *d_akx,hipfftDoubleComplex *d_aky,hipfftDoubleComplex *d_akz,hipfftDoubleComplex shix,hipfftDoubleComplex shiy,hipfftDoubleComplex shiz,int nxyz)
{
	unsigned int ind = blockIdx.x*blockDim.x+threadIdx.x;
	hipfftDoubleReal SAVE2;

	if (ind<nxyz)
	{
                shix.x = shix.x*d_akx[ind].x;
		shiy.x = shiy.x*d_aky[ind].x;
		shiz.x = shiz.x*d_akz[ind].x;
                shix.y = shix.y*d_akx[ind].y;
		shiy.y = shiy.y*d_aky[ind].y;
		shiz.y = shiz.y*d_akz[ind].y;

		//multiplication by shix
		SAVE2         = exp(shix.x)*(d_ffta[ind].x*cos(shix.y)-sin(shix.y)*d_ffta[ind].y);
		d_ffta[ind].y = exp(shix.x)*(d_ffta[ind].x*sin(shix.y)+cos(shix.y)*d_ffta[ind].y);
		d_ffta[ind].x = SAVE2;
		//multiplication by shiy
		SAVE2         = exp(shiy.x)*(d_ffta[ind].x*cos(shiy.y)-sin(shiy.y)*d_ffta[ind].y);
		d_ffta[ind].y = exp(shiy.x)*(d_ffta[ind].x*sin(shiy.y)+cos(shiy.y)*d_ffta[ind].y);
		d_ffta[ind].x = SAVE2;
		//multiplication by shiz
		SAVE2         = exp(shiz.x)*(d_ffta[ind].x*cos(shiz.y)-sin(shiz.y)*d_ffta[ind].y);
		d_ffta[ind].y = exp(shiz.x)*(d_ffta[ind].x*sin(shiz.y)+cos(shiz.y)*d_ffta[ind].y);
		d_ffta[ind].x = SAVE2;
	}
}

extern "C" void multiply_shift_(hipfftDoubleComplex *d_ffta,hipfftDoubleComplex *d_akx,hipfftDoubleComplex *d_aky,hipfftDoubleComplex *d_akz,double *sx,double *sy,double *sz,int *N)
{
	int nxyz = *N;
	hipfftDoubleComplex shix,shiy,shiz;
	int blocksize=192;
	int gridx=(int)ceil(nxyz/(float)blocksize);
	dim3 dimgrid(gridx,1,1);
	dim3 dimblock(blocksize,1,1);

	shix.x = *sx;
	shix.y = *sx;
	shiy.x = *sy;
	shiy.y = *sy;
	shiz.x = *sz;
	shiz.y = *sz;

	//Multiplication on the GPU
        multiply_shift_gpu<<<dimgrid,dimblock,0,stream2>>>(d_ffta,d_akx,d_aky,d_akz,shix,shiy,shiz,nxyz);
	Check_CUDA_Error(error);
}

__global__ void hpsi_gpu(hipfftDoubleComplex *d_ffta,hipfftDoubleComplex *d_ffta2,hipfftDoubleReal *d_akv,int nxyz)
{
	unsigned int ind = blockIdx.x*blockDim.x+threadIdx.x;

	if (ind<nxyz)
	{
		d_ffta2[ind].x = d_akv[ind]*d_ffta[ind].x+d_ffta2[ind].x;
		d_ffta2[ind].y = d_akv[ind]*d_ffta[ind].y+d_ffta2[ind].y;
	}
}

extern "C" void hpsi_cuda_(hipfftDoubleComplex *d_ffta,hipfftDoubleComplex *d_ffta2,hipfftDoubleReal *d_akv,int *N)
{
	int nxyz = *N;
	int blocksize=192;
	int gridx=(int)ceil(nxyz/(float)blocksize);
	dim3 dimgrid(gridx,1,1);
	dim3 dimblock(blocksize,1,1);

	//d_ffta*d_ak+d_ffta2 on the GPU
        hpsi_gpu<<<dimgrid,dimblock,0,stream2>>>(d_ffta,d_ffta2,d_akv,nxyz);
	Check_CUDA_Error(error);
}

__global__ void d_grad_gpu1(hipfftDoubleComplex *d_ffta,hipfftDoubleComplex *d_ffta2,hipfftDoubleReal *d_akv,double epswf,double e0dmp,int nxyz)
{
	unsigned int ind = blockIdx.x*blockDim.x+threadIdx.x;

	if (ind<nxyz)
	{
		d_ffta[ind].x = d_ffta[ind].x-((epswf*d_ffta2[ind].x)/(d_akv[ind]+e0dmp));
		d_ffta[ind].y = d_ffta[ind].y-((epswf*d_ffta2[ind].y)/(d_akv[ind]+e0dmp));
	}
}

extern "C" void d_grad1_(hipfftDoubleComplex *d_ffta,hipfftDoubleComplex *d_ffta2,hipfftDoubleReal *d_akv,double *ep,double *e0,int *N)
{
	int nxyz = *N;
	double epswf = *ep, e0dmp = *e0;
	int blocksize=192;
	int gridx=(int)ceil(nxyz/(float)blocksize);
	dim3 dimgrid(gridx,1,1);
	dim3 dimblock(blocksize,1,1);

	//grad on the GPU
        d_grad_gpu1<<<dimgrid,dimblock,0,stream2>>>(d_ffta,d_ffta2,d_akv,epswf,e0dmp,nxyz);
	Check_CUDA_Error(error);
}

__global__ void d_grad_gpu2(hipfftDoubleComplex *d_ffta,hipfftDoubleComplex *d_ffta2,double epswf,int nxyz)
{
	unsigned int ind = blockIdx.x*blockDim.x+threadIdx.x;

	if (ind<nxyz)
	{
		d_ffta[ind].x = d_ffta[ind].x-epswf*d_ffta2[ind].x;
		d_ffta[ind].y = d_ffta[ind].y-epswf*d_ffta2[ind].y;
	}
}

extern "C" void d_grad2_(hipfftDoubleComplex *d_ffta,hipfftDoubleComplex *d_ffta2,double *ep,int *N)
{
	int nxyz = *N;
	double epswf = *ep;
	int blocksize=192;
	int gridx=(int)ceil(nxyz/(float)blocksize);
	dim3 dimgrid(gridx,1,1);
	dim3 dimblock(blocksize,1,1);

	//grad on the GPU
        d_grad_gpu2<<<dimgrid,dimblock,0,stream2>>>(d_ffta,d_ffta2,epswf,nxyz);
	Check_CUDA_Error(error);
}

__global__ void d_sum_calc(hipfftDoubleComplex *d_ffta,hipfftDoubleComplex *d_ffta2,hipfftDoubleReal *d_akv,double *d_sum0,double *d_sumk,double *d_sume,double *d_sum2,int nxyz)
{
	unsigned int ind = blockIdx.x*blockDim.x+threadIdx.x;

	if (ind<nxyz)
	{
		d_sum0[ind] = d_ffta[ind].x*d_ffta[ind].x + d_ffta[ind].y*d_ffta[ind].y;
		d_sumk[ind] = d_sum0[ind]*d_akv[ind];
		d_sume[ind] = d_ffta2[ind].x*d_ffta[ind].x + d_ffta2[ind].y*d_ffta[ind].y;
		d_sum2[ind] = d_ffta2[ind].x*d_ffta2[ind].x + d_ffta2[ind].y*d_ffta2[ind].y;
	}
}

extern "C" void sum_calc_(double *s0,double *sk,double *se,double *s2,hipfftDoubleComplex *d_ffta,hipfftDoubleComplex *d_ffta2, hipfftDoubleReal *d_akv,int *N)
{
	int nxyz = *N;
	int blocksize=192;
	int gridx=(int)ceil(nxyz/(float)blocksize);
	dim3 dimgrid(gridx,1,1);
	dim3 dimblock(blocksize,1,1);
	//Declaration of the vectors on the GPU
	thrust::device_vector<double> d_sum0(nxyz);
	thrust::device_vector<double> d_sumk(nxyz);
	thrust::device_vector<double> d_sume(nxyz);
	thrust::device_vector<double> d_sum2(nxyz);

	//Computation of the d_sum vectors on the GPU
        d_sum_calc<<<dimgrid,dimblock,0,stream2>>>(d_ffta,d_ffta2,d_akv,raw_pointer_cast(&d_sum0[0]),raw_pointer_cast(&d_sumk[0]),raw_pointer_cast(&d_sume[0]),raw_pointer_cast(&d_sum2[0]),nxyz);
	Check_CUDA_Error(error);
	//Reduction of the vectors
	*s0=thrust::reduce(d_sum0.begin(),d_sum0.end(),(double)0.0);
	*sk=thrust::reduce(d_sumk.begin(),d_sumk.end(),(double)0.0);
	*se=thrust::reduce(d_sume.begin(),d_sume.end(),(double)0.0);
	*s2=thrust::reduce(d_sum2.begin(),d_sum2.end(),(double)0.0);
}
